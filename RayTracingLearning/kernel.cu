// kernel.cu
#include <hip/hip_runtime.h>
#include <stdint.h> // For uint32_t

__global__ void renderKernel(uint32_t* imageData, int width, int height, uint32_t color)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int index = y * width + x;
        imageData[index] = color;
    }
}
